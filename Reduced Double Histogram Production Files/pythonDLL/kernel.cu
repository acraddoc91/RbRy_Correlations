#include "hip/hip_runtime.h"
//  Microsoft
#define DLLEXPORT extern "C" __declspec(dllexport)

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <iostream>
#include <vector>
#include <map>

#include "H5Cpp.h"
#include <H5Exception.h>

#include <Python.h>

#include <omp.h>

//Define integers explicitly to prevent problems on different platforms
#define int8 __int8
#define int16 __int16
#define int32 __int32
#define int64 __int64

const int max_tags_length = 200000;
const int max_clock_tags_length = 5000;
const int max_channels = 3;
const size_t return_size = 3;
const int file_block_size = 16;
const double tagger_resolution = 82.3e-12;
const int num_gpu = 1;
const int threads_per_cuda_block_numer = 64;
const int shared_mem_size = 4;

struct shotData {
	bool file_load_completed;
	std::vector<int16> channel_list;
	std::map<int16, int16> channel_map;
	std::vector<int64> start_tags;
	std::vector<int64> end_tags;
	std::vector<int64> photon_tags;
	std::vector<int64> clock_tags;
	std::vector<std::vector<int64>> sorted_photon_tags;
	std::vector<std::vector<int32>> sorted_photon_bins;
	std::vector<std::vector<int64>> sorted_clock_tags;
	std::vector<std::vector<int32>> sorted_clock_bins;
	std::vector<int32> sorted_photon_tag_pointers;
	std::vector<int32> sorted_clock_tag_pointers;

	shotData() : sorted_photon_tags(max_channels, std::vector<int64>(max_tags_length, 0)), sorted_photon_bins(max_channels, std::vector<int32>(max_tags_length, 0)), sorted_photon_tag_pointers(max_channels, 0), sorted_clock_tags(2, std::vector<int64>(max_clock_tags_length, 0)), sorted_clock_bins(2, std::vector<int32>(max_clock_tags_length, 0)), sorted_clock_tag_pointers(2, 0) {}
};

struct gpuData {
	int32 *coinc_gpu;
	int32 *photon_bins_gpu;
	int32 *start_and_end_clocks_gpu;
	int *max_bin_gpu, *pulse_spacing_gpu, *max_pulse_distance_gpu, *photon_bins_length_gpu;
	int *offset_gpu;
};

__global__ void calculateCoincidencesGPU_g2_nominibatching(int32 *coinc, int32 *photon_bins, int32 *start_and_end_clocks, int *max_bin, int *pulse_spacing, int *max_pulse_distance, int *offset, int *photon_bins_length, int num_channels, int shot_file_num) {
	//Get numerator step to work on
	int id = threadIdx.x;
	int block = blockIdx.x;
	int block_size = blockDim.x;

	//Check we're not calculating something out of range
	if (block * block_size + id < ((*max_bin * 2 + 1) + (*max_pulse_distance * 2))) {
		int pulse_shift_measurment = (block * block_size + id >= *max_bin * 2 + 1) && (block * block_size + id < *max_bin * 2 + 1 + (*max_pulse_distance * 2));
		int pulse_shift = ((block * block_size + id - (*max_bin * 2 + 1) - (*max_pulse_distance)) + ((block * block_size + id - (*max_bin * 2 + 1) - (*max_pulse_distance)) >= 0)) * (pulse_shift_measurment);
		int tau = (block * block_size + id - (*max_bin)) * (!pulse_shift_measurment);
		tau += pulse_shift * (*pulse_spacing);
		for (int channel_1 = 0; channel_1 < num_channels; channel_1++) {
			for (int channel_2 = channel_1 + 1; channel_2 < num_channels; channel_2++) {
				int i = 0;
				int j = 0;
				int running_tot = 0;
				while ((i < photon_bins_length[channel_1 + shot_file_num * max_channels]) && (j < photon_bins_length[channel_2 + shot_file_num * max_channels])) {
					//Check if we're outside the window of interest
					int out_window = (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] < (*max_bin + *max_pulse_distance * *pulse_spacing + start_and_end_clocks[0 + shot_file_num * 2])) || (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] > (start_and_end_clocks[1 + shot_file_num * 2] - (*max_bin + *max_pulse_distance * *pulse_spacing)));
					//Increment i if chan_1 < chan_2
					int c1gc2 = !out_window && (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] < (photon_bins[offset[channel_2 + shot_file_num * max_channels] + j] - tau));
					//Check if we have a common element increment
					int c1ec2 = !out_window && (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] == (photon_bins[offset[channel_2 + shot_file_num * max_channels] + j] - tau));
					//Increment running total if channel 1 equals channel 2
					running_tot += c1ec2;
					//Increment channel 1 if it is greater than channel 2, equal to channel 2 or ouside of the window
					i += (c1gc2 + c1ec2 + out_window);
					j += !(c1gc2 + c1ec2 + out_window) + c1ec2;
				}
				coinc[block * block_size + id + shot_file_num * ((*max_bin * 2 + 1) + (*max_pulse_distance * 2))] += running_tot;
			}
		}
	}
}

__global__ void calculateCoincidencesGPU_g2(int32 *coinc, int32 *photon_bins, int32 *start_and_end_clocks, int32 *max_bin, int32 *pulse_spacing, int32 *max_pulse_distance, int32 *offset, int32 *photon_bins_length, int8 num_channels, int32 shot_file_num) {
	//Get numerator step to work on
	int16 id = threadIdx.x;
	int32 block = blockIdx.x;
	int16 block_size = blockDim.x;

	int32 max_bin_gpu = *max_bin;
	int32 pulse_spacing_gpu = *pulse_spacing;
	int32 max_pulse_distance_gpu = *max_pulse_distance;
	int8 num_channels_gpu = num_channels;
	int32 shot_file_num_gpu = shot_file_num;
	__shared__ int32 photons_bins_a[shared_mem_size * threads_per_cuda_block_numer];
	__shared__ int32 photons_bins_b[shared_mem_size * threads_per_cuda_block_numer];

	//Check we're not calculating something out of range
	if (block * block_size + id < ((max_bin_gpu * 2 + 1) + (max_pulse_distance_gpu * 2))) {
		int32 pulse_shift_measurment = (block * block_size + id >= max_bin_gpu * 2 + 1) && (block * block_size + id < max_bin_gpu * 2 + 1 + (max_pulse_distance_gpu * 2));
		int32 pulse_shift = ((block * block_size + id - (max_bin_gpu * 2 + 1) - (max_pulse_distance_gpu)) + ((block * block_size + id - (max_bin_gpu * 2 + 1) - (max_pulse_distance_gpu)) >= 0)) * (pulse_shift_measurment);
		int32 tau = (block * block_size + id - (max_bin_gpu)) * (!pulse_shift_measurment);
		int32 start_clock = start_and_end_clocks[0 + shot_file_num * 2];
		int32 end_clock = start_and_end_clocks[1 + shot_file_num * 2];
		
		tau += pulse_shift * (pulse_spacing_gpu);
		for (int8 channel_1 = 0; channel_1 < num_channels_gpu; channel_1++) {
			for (int8 channel_2 = channel_1 + 1; channel_2 < num_channels_gpu; channel_2++) {
				int32 i = 0;
				int32 j = 0;
				int32 l_a = photon_bins_length[channel_1 + shot_file_num_gpu * max_channels];
				int32 l_b = photon_bins_length[channel_2 + shot_file_num_gpu * max_channels];
				int32 running_tot = 0;
				while ((i < l_a) && (j < l_b)) {
					int8 i_a = 0;
					int8 j_b = 0;
					int8 out_window;
					int8 c1gc2;
					int8 c1ec2;
					//Load up from global to shared memory some bins
					#pragma unroll
					for (int8 dummy = 0; dummy < shared_mem_size; dummy++) {
						photons_bins_a[dummy + id * shared_mem_size] = photon_bins[offset[channel_1 + shot_file_num_gpu * max_channels] + i + dummy];
					}
					#pragma unroll
					for (int8 dummy = 0; dummy < shared_mem_size; dummy++) {
						photons_bins_b[dummy + id * shared_mem_size] = photon_bins[offset[channel_2 + shot_file_num_gpu * max_channels] + j + dummy];
					}
					while ((i_a < shared_mem_size) && (j_b < shared_mem_size) && (i_a + i < l_a) && (j_b + j < l_b)) {
						//Check if we're outside the window of interest
						out_window = (photons_bins_a[i_a + id * shared_mem_size] < (max_bin_gpu + max_pulse_distance_gpu * pulse_spacing_gpu + start_clock)) || (photons_bins_a[i_a + id * shared_mem_size] > (end_clock - (max_bin_gpu + max_pulse_distance_gpu * pulse_spacing_gpu)));
						//Increment j if chan_1 > chan_2
						c1gc2 = (photons_bins_a[i_a + id * shared_mem_size] > (photons_bins_b[j_b + id * shared_mem_size] - tau));
						//Check if we have a common element increment
						c1ec2 = (photons_bins_a[i_a + id * shared_mem_size] == (photons_bins_b[j_b + id * shared_mem_size] - tau));
						//Increment running total if channel 1 equals channel 2
						running_tot += !(out_window) && c1ec2;
						//Increment channel 1 if it is greater than channel 2, equal to channel 2 or ouside of the window
						i_a += (!c1gc2 || out_window);
						j_b += (c1gc2 || c1ec2);
					}
					i += i_a;
					j += j_b;
				}
				coinc[block * block_size + id + shot_file_num_gpu * ((max_bin_gpu * 2 + 1) + (max_pulse_distance_gpu * 2))] += running_tot;
			}
		}
	}
}

void calculateCoincidencesGPU_g2_cpu(shotData *shot_data, int32 id, int32 *max_bin, int32 *pulse_spacing, int32 *max_pulse_distance, int32 *coinc, int32 shot_file_num) {
	
	if (id < ((*max_bin * 2 + 1) + (*max_pulse_distance * 2))) {
		int32 pulse_shift_measurment = (id >= *max_bin * 2 + 1) && (id < *max_bin * 2 + 1 + (*max_pulse_distance * 2));
		int32 pulse_shift = ((id - (*max_bin * 2 + 1) - (*max_pulse_distance)) + ((id - (*max_bin * 2 + 1) - (*max_pulse_distance)) >= 0)) * (pulse_shift_measurment);
		int32 tau = (id - (*max_bin)) * (!pulse_shift_measurment);
		tau += pulse_shift * (*pulse_spacing);

		int32 start_clock = shot_data->sorted_clock_bins[1][0];
		int32 end_clock = shot_data->sorted_clock_bins[0][0];

		for (int channel_1 = 0; channel_1 < shot_data->channel_list.size(); channel_1++) {
			for (int channel_2 = channel_1 + 1; channel_2 < shot_data->channel_list.size(); channel_2++) {

				int i = 0;
				int j = 0;
				int running_tot = 0;

				while ((i < shot_data->sorted_photon_tag_pointers[channel_1]) && (j < shot_data->sorted_photon_tag_pointers[channel_2])) {

					//Check if we're outside the window of interest
					int out_window = (shot_data->sorted_photon_bins[channel_1][i] < (*max_bin + *max_pulse_distance * *pulse_spacing + start_clock)) || (shot_data->sorted_photon_bins[channel_1][i] > (end_clock - (*max_bin + *max_pulse_distance * *pulse_spacing)));
					//chan_1 > chan_2
					int c1gc2 = shot_data->sorted_photon_bins[channel_1][i] > (shot_data->sorted_photon_bins[channel_2][j] - tau);
					//Check if we have a common element increment
					int c1ec2 = shot_data->sorted_photon_bins[channel_1][i] == (shot_data->sorted_photon_bins[channel_2][j] - tau);
					//Increment running total if channel 1 equals channel 2
					running_tot += !out_window && c1ec2;
					//Increment channel 1 if it is greater than channel 2, equal to channel 2 or ouside of the window
					i += (!c1gc2 || out_window);
					j += (c1gc2 || c1ec2);

				}
				coinc[id + shot_file_num * ((*max_bin * 2 + 1) + (*max_pulse_distance * 2))] += running_tot;
				//coinc[id + shot_file_num * ((*max_bin * 2 + 1) + (*max_pulse_distance * 2))] = tau;
			}
		}
	}
}

//Function grabs all tags and channel list from file
void fileToShotData(shotData *shot_data, char* filename) {
	//Open up file
	H5::H5File file(filename, H5F_ACC_RDONLY);
	//Open up "Tags" group
	H5::Group tag_group(file.openGroup("Tags"));
	//Find out how many tag sets there are, should be 4 if not something is fucky
	hsize_t numTagsSets = tag_group.getNumObjs();
	if (numTagsSets != 4) {
		printf("There should be 4 sets of Tags, found %i\n", numTagsSets);
		delete filename;
		exit;
	}
	//Read tags to shotData structure
	//First the clock tags
	H5::DataSet clock_dset(tag_group.openDataSet("ClockTags0"));
	H5::DataSpace clock_dspace = clock_dset.getSpace();
	hsize_t clock_length[1];
	clock_dspace.getSimpleExtentDims(clock_length, NULL);
	shot_data->clock_tags.resize(clock_length[0]);
	clock_dset.read(&(*shot_data).clock_tags[0u], H5::PredType::NATIVE_UINT64, clock_dspace);
	clock_dspace.close();
	clock_dset.close();
	//Then start tags
	H5::DataSet start_dset(tag_group.openDataSet("StartTag"));
	H5::DataSpace start_dspace = start_dset.getSpace();
	hsize_t start_length[1];
	start_dspace.getSimpleExtentDims(start_length, NULL);
	shot_data->start_tags.resize(start_length[0]);
	start_dset.read(&(*shot_data).start_tags[0u], H5::PredType::NATIVE_UINT64, start_dspace);
	start_dspace.close();
	start_dset.close();
	//Then end tags
	H5::DataSet end_dset(tag_group.openDataSet("EndTag"));
	H5::DataSpace end_dspace = end_dset.getSpace();
	hsize_t end_length[1];
	end_dspace.getSimpleExtentDims(end_length, NULL);
	shot_data->end_tags.resize(end_length[0]);
	end_dset.read(&(*shot_data).end_tags[0u], H5::PredType::NATIVE_UINT64, end_dspace);
	end_dspace.close();
	end_dset.close();
	//Finally photon tags
	H5::DataSet photon_dset(tag_group.openDataSet("TagWindow0"));
	H5::DataSpace photon_dspace = photon_dset.getSpace();
	hsize_t photon_length[1];
	photon_dspace.getSimpleExtentDims(photon_length, NULL);
	shot_data->photon_tags.resize(photon_length[0]);
	photon_dset.read(&(*shot_data).photon_tags[0u], H5::PredType::NATIVE_UINT64, photon_dspace);
	photon_dspace.close();
	photon_dset.close();
	//And close tags group
	tag_group.close();
	//Open up "Inform" group
	H5::Group inform_group(file.openGroup("Inform"));
	//Grab channel list
	H5::DataSet chan_dset(inform_group.openDataSet("ChannelList"));
	H5::DataSpace chan_dspace = chan_dset.getSpace();
	hsize_t chan_length[1];
	chan_dspace.getSimpleExtentDims(chan_length, NULL);
	shot_data->channel_list.resize(chan_length[0]);
	chan_dset.read(&(*shot_data).channel_list[0u], H5::PredType::NATIVE_UINT16, chan_dspace);
	chan_dspace.close();
	chan_dset.close();
	//Close Inform group
	inform_group.close();
	//Close file
	file.close();

	//Populate channel map
	for (int16 i = 0; i < shot_data->channel_list.size(); i++) {
		shot_data->channel_map[shot_data->channel_list[i]] = i;
	}
}

//Reads relevant information for a block of files into shot_block
void populateBlock(std::vector<shotData> *shot_block, std::vector<char *> *filelist, int block_num, int num_devices, int block_size) {
	//Loop over the block size
	for (int i = 0; i < block_size * num_devices; i++) {
		//Default to assuming the block is corrupted
		(*shot_block)[i].file_load_completed = false;
		//Figure out the file id within the filelist
		int file_id = block_num * block_size * num_devices + i;
		//Check the file_id isn't out of range of the filelist
		if (file_id < filelist->size()) {
			//Try to load file to shot_block
			try {
				fileToShotData(&(*shot_block)[i], (*filelist)[file_id]);
				(*shot_block)[i].file_load_completed = true;
			}
			//Will catch if the file is corrupted, print corrupted filenames to command window
			catch (...) {
				printf("%s appears corrupted\n", (*filelist)[file_id]);
			}
		}
	}
}

//Process the time tags, assigning them to the correct channel, binning them appropriately and removing tags which do not fall in the clock mask
void sortTags(shotData *shot_data) {
	int32 i;
	int high_count = 0;
	//Loop over all tags in clock_tags
	for (i = 0; i < shot_data->clock_tags.size(); i++) {
		//Check if clock tag is a high word
		if (shot_data->clock_tags[i] & 1) {
			//Up the high count
			high_count++;
		}
		else {
			//Determine whether it is the rising (start) or falling (end) slope
			int slope = ((shot_data->clock_tags[i] >> 28) & 1);
			//Put tag in appropriate clock tag vector and increment the pointer for said vector
			shot_data->sorted_clock_tags[slope][shot_data->sorted_clock_tag_pointers[slope]] = ((shot_data->clock_tags[i] >> 1) & 0x7FFFFFF) + (high_count << 27) - ((shot_data->start_tags[1] >> 1) & 0x7FFFFFF);
			shot_data->sorted_clock_tag_pointers[slope]++;
		}
	}
	high_count = 0;
	//Clock pointer
	int clock_pointer = 0;
	//Loop over all tags in photon_tags
	for (i = 0; i < shot_data->photon_tags.size(); i++) {
		//Check if photon tag is a high word
		if (shot_data->photon_tags[i] & 1) {
			//Up the high count
			high_count++;
		}
		else {
			//Figure out if it fits within the mask
			int64 time_tag = ((shot_data->photon_tags[i] >> 1) & 0x7FFFFFF) + (high_count << 27) - ((shot_data->start_tags[1] >> 1) & 0x7FFFFFF);
			bool valid = true;
			while (valid) {
				//printf("%i\t%i\t%i\t", time_tag, shot_data->sorted_clock_tags[1][clock_pointer], shot_data->sorted_clock_tags[0][clock_pointer - 1]);
				//Increment dummy pointer if channel tag is greater than current start tag
				if ((time_tag >= shot_data->sorted_clock_tags[1][clock_pointer]) & (clock_pointer < shot_data->sorted_clock_tag_pointers[1])) {
					//printf("up clock pointer\n");
					clock_pointer++;
				}
				//Make sure clock_pointer is greater than 0, preventing an underflow error
				else if (clock_pointer > 0) {
					//Check if tag is lower than previous end tag i.e. startTags[j-1] < channeltags[i] < endTags[j-1]
					if (time_tag <= shot_data->sorted_clock_tags[0][clock_pointer - 1]) {
						//printf("add tag tot data\n");
						//Determine the index for given tag
						int channel_index;
						//Bin tag and assign to appropriate vector
						channel_index = shot_data->channel_map.find(((shot_data->photon_tags[i] >> 29) & 7) + 1)->second;
						shot_data->sorted_photon_tags[channel_index][shot_data->sorted_photon_tag_pointers[channel_index]] = time_tag;
						shot_data->sorted_photon_tag_pointers[channel_index]++;
						//printf("%i\t%i\t%i\n", channel_index, time_tag, shot_data->sorted_photon_tag_pointers[channel_index]);
					}
					//Break the valid loop
					valid = false;
				}
				// If tag is smaller than the first start tag
				else {
					valid = false;
				}
			}
		}
	}
}

//Converts our tags to bins with a given bin width
void tagsToBins(shotData *shot_data, double bin_width) {
	int tagger_bins_per_bin_width = (int)round(bin_width / tagger_resolution);
#pragma omp parallel for
	for (int channel = 0; channel < shot_data->sorted_photon_bins.size(); channel++) {
#pragma omp parallel for
		for (int i = 0; i < shot_data->sorted_photon_tag_pointers[channel]; i++) {
			shot_data->sorted_photon_bins[channel][i] = shot_data->sorted_photon_tags[channel][i] / tagger_bins_per_bin_width;
		}
	}
	for (int slope = 0; slope <= 1; slope++) {
#pragma omp parallel for
		for (int i = 0; i < shot_data->sorted_clock_tag_pointers[slope]; i++) {
			shot_data->sorted_clock_bins[slope][i] = shot_data->sorted_clock_tags[slope][i] / tagger_bins_per_bin_width;
		}
	}
}

//Sorts photons and bins them for each file in a block
void sortAndBinBlock(std::vector<shotData> *shot_block, double bin_width, int num_devices, int block_size) {
#pragma omp parallel for
	for (int shot_file_num = 0; shot_file_num < (block_size * num_devices); shot_file_num++) {
		if ((*shot_block)[shot_file_num].file_load_completed) {
			sortTags(&(*shot_block)[shot_file_num]);
			tagsToBins(&(*shot_block)[shot_file_num], bin_width);
		}
	}
}

void printShotChannelBins(shotData *shot_data, int channel) {
	for (int i = 0; i < shot_data->sorted_photon_tag_pointers[channel]; i++) {
		printf("%i\t%i\t%i\n", i, shot_data->sorted_photon_tags[channel][i], shot_data->sorted_photon_bins[channel][i]);
	}
}

DLLEXPORT void count_tags(char **file_list, int file_list_length, double max_time, double bin_width, double pulse_spacing, int max_pulse_distance) {
	std::vector<char *> filelist(file_list_length);
	//Grab filename and stick it into filelist vector
	for (int i = 0; i < file_list_length; i++) {
		filelist[i] = file_list[i];
	}

	int blocks_req;
	if (file_list_length< (file_block_size * 1)) {
		blocks_req = 1;
	}
	else if ((file_list_length % (file_block_size * 1)) == 0) {
		blocks_req = file_list_length / (file_block_size * 1);
	}
	else {
		blocks_req = file_list_length / (file_block_size * 1) + 1;
	}

	int max_bin = (int)round(max_time / bin_width);
	int bin_pulse_spacing = (int)round(pulse_spacing / bin_width);

	int num_tags_split[file_block_size][3] = { 0 };
	int num_tags_tot[3] = { 0 };

	printf("Max Time\tBin Width\tPulse Spacing\tMax Pulse Distance\n");
	printf("%fus\t%fns\t%fus\t%i\n", max_time * 1e6, bin_width * 1e9, pulse_spacing * 1e6, max_pulse_distance);

	//Processes files in blocks
	for (int block_num = 0; block_num < blocks_req; block_num++) {
		//Allocate a vector to hold a block of shot_data
		std::vector<shotData> shot_block(file_block_size * 1);

		//Populate the shot_block with data from file
		populateBlock(&shot_block, &filelist, block_num, 1, file_block_size);

		//Sort tags and convert them to bins
		sortAndBinBlock(&shot_block, bin_width, 1, file_block_size);

		for (int shot_file_num = 0; shot_file_num < file_block_size; shot_file_num++) {
			if ((shot_block)[shot_file_num].file_load_completed) {
				for (int channel = 0; channel < 3; channel++) {
					num_tags_tot[channel] += shot_block[shot_file_num].sorted_photon_tag_pointers[channel];
				}
			}
		}

		#pragma omp parallel for
		for (int shot_file_num = 0; shot_file_num < file_block_size; shot_file_num++) {
			if ((shot_block)[shot_file_num].file_load_completed) {
				int shot_num_tags[3] = { 0 };
				//printf("%f,%f\n", (shot_block[shot_file_num].sorted_clock_bins[1][0] + max_bin + bin_pulse_spacing * max_pulse_distance) * bin_width, (shot_block[shot_file_num].sorted_clock_bins[0][0] - max_bin - bin_pulse_spacing * max_pulse_distance) * bin_width);
				#pragma omp parallel for
				for (int channel = 0; channel < 3; channel++) {
					for (int i = 0; i < shot_block[shot_file_num].sorted_photon_tag_pointers[channel]; i++) {
						shot_num_tags[channel] += (shot_block[shot_file_num].sorted_photon_bins[channel][i] > shot_block[shot_file_num].sorted_clock_bins[1][0] + max_bin + bin_pulse_spacing * max_pulse_distance) && (shot_block[shot_file_num].sorted_photon_bins[channel][i] < shot_block[shot_file_num].sorted_clock_bins[0][0] - max_bin - bin_pulse_spacing * max_pulse_distance);
					}
					num_tags_split[shot_file_num][channel] += shot_num_tags[channel];
				}
			}
		}
	}
	int num_tags[3] = { 0 };
	for (int shot_file_num = 0; shot_file_num < file_block_size; shot_file_num++) {
		for (int channel = 0; channel < 3; channel++) {
			num_tags[channel] += num_tags_split[shot_file_num][channel];
		}
	}
	for (int channel = 0; channel < 3; channel++) {
		printf("Channel %i has %i out of %i tags in window\n", channel, num_tags[channel], num_tags_tot[channel]);
	}
}

DLLEXPORT void getG2Correlations(char **file_list, int file_list_length, double max_time, double bin_width, double pulse_spacing, int max_pulse_distance, PyObject *numer, int32 *denom) {
	
	std::vector<char *> filelist(file_list_length);
	//Grab filename and stick it into filelist vector
	for (int i = 0; i < file_list_length; i++) {
		filelist[i] = file_list[i];
	}

	int blocks_req;
	if (file_list_length< (file_block_size * num_gpu)) {
		blocks_req = 1;
	}
	else if ((file_list_length % (file_block_size * num_gpu)) == 0) {
		blocks_req = file_list_length / (file_block_size * num_gpu);
	}
	else {
		blocks_req = file_list_length / (file_block_size * num_gpu) + 1;
	}
	printf("Chunking %i files into %i blocks\n", file_list_length, blocks_req);
	printf("Max Time\tBin Width\tPulse Spacing\tMax Pulse Distance\n");
	printf("%fus\t%fns\t%fus\t%i\n", max_time * 1e6, bin_width * 1e9, pulse_spacing * 1e6, max_pulse_distance);

	int max_bin = (int)round(max_time / bin_width);
	int bin_pulse_spacing = (int)round(pulse_spacing / bin_width);

	//Figure out how many blocks to chunk the processing up into
	//For the numerator
	int cuda_blocks_numer = 0;
	if (threads_per_cuda_block_numer >= (max_bin * 2 + 1) + (max_pulse_distance * 2)) {
		cuda_blocks_numer = 1;
	}
	else if (((max_bin * 2 + 1) % threads_per_cuda_block_numer) == 0) {
		cuda_blocks_numer = ((max_bin * 2 + 1) + (max_pulse_distance * 2)) / threads_per_cuda_block_numer;
	}
	else {
		cuda_blocks_numer = ((max_bin * 2 + 1) + (max_pulse_distance * 2)) / threads_per_cuda_block_numer + 1;
	}

	hipError_t cudaStatus;

	//Pointers for our various pinned memory for host-GPU DMA
	int32* pinned_photon_bins[num_gpu];
	int32* pinned_start_and_end_clocks[num_gpu];
	int32* pinned_photon_bins_length[num_gpu];

	//Load some stuff to the GPU we will use permenantly
	//Allocate memory on GPU for various things
	gpuData gpu_data[num_gpu];
	
	for (int gpu = 0; gpu < num_gpu; gpu++) {

		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		hipHostMalloc((int32**)&pinned_photon_bins[gpu], max_tags_length * max_channels * file_block_size * sizeof(int32));
		hipHostMalloc((int32**)&pinned_start_and_end_clocks[gpu], 2 * file_block_size * sizeof(int32));
		hipHostMalloc((int32**)&pinned_photon_bins_length[gpu], max_channels * file_block_size * sizeof(int32));

		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).photon_bins_gpu), max_channels * max_tags_length * file_block_size * sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc photon_bins_gpu failed\n");
			printf("%s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).offset_gpu), max_channels * file_block_size * sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc offset_gpu failed!\n");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).photon_bins_length_gpu), max_channels * file_block_size * sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc photon_bins_length_gpu failed!\n");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).coinc_gpu), ((2 * (max_bin)+1) + (max_pulse_distance * 2)) * file_block_size * sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc numer_gpu failed!\n");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).start_and_end_clocks_gpu), 2 * file_block_size * sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc start_and_end_clocks_gpu failed!\n");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).max_bin_gpu), sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc max_bin_gpu failed!\n");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).pulse_spacing_gpu), sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc pulse_spacing_gpu failed!\n");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).max_pulse_distance_gpu), sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc max_pulse_distance_gpu failed!\n");
			goto Error;
		}

		//And set some values that are constant across all data
		cudaStatus = hipMemcpy(((gpu_data[gpu]).max_bin_gpu), &max_bin, sizeof(int32), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			printf("hipMemcpy failed!\n");
			goto Error;
		}
		cudaStatus = hipMemcpy(((gpu_data[gpu]).pulse_spacing_gpu), &bin_pulse_spacing, sizeof(int32), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			printf("hipMemcpy failed!\n");
			goto Error;
		}
		cudaStatus = hipMemcpy(((gpu_data[gpu]).max_pulse_distance_gpu), &max_pulse_distance, sizeof(int32), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			printf("hipMemcpy failed!\n");
			goto Error;
		}

		//Pointer to first photon bin for each channel
		int host_offest_array[max_channels * file_block_size];
		for (int i = 0; i < max_channels * file_block_size; i++) {
			host_offest_array[i] = i * max_tags_length;
		}
		cudaStatus = hipMemcpy(((gpu_data[gpu]).offset_gpu), host_offest_array, max_channels * file_block_size * sizeof(int32), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			printf("hipMemcpy failed!\n");
			goto Error;
		}

		//Set numerator and denominator to 0
		cudaStatus = hipMemset(((gpu_data[gpu])).coinc_gpu, 0, ((2 * (max_bin)+1) + (max_pulse_distance * 2)) * file_block_size * sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMemset failed!\n");
			goto Error;
		}
	}

	//Create some streams for us to use for GPU parallelism
	hipStream_t streams[num_gpu][file_block_size];
	for (int gpu = 0; gpu < num_gpu; gpu++) {
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}
		for (int i = 0; i < file_block_size; i++) {
			cudaStatus = hipStreamCreate(&streams[gpu][i]);
			if (cudaStatus != hipSuccess) {
				printf("Failed to create streams %s\n", hipGetErrorString(cudaStatus));
				goto Error;
			}
		}
	}

	//Create some events to allow us to know if previous transfer has completed
	hipEvent_t events[num_gpu][file_block_size];
	for (int gpu = 0; gpu < num_gpu; gpu++) {
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}
		for (int i = 0; i < file_block_size; i++) {
			hipEventCreate(&events[gpu][i]);
			if (cudaStatus != hipSuccess) {
				printf("Failed to create events %s\n", hipGetErrorString(cudaStatus));
				goto Error;
			}
		}
	}

	//Processes files in blocks
	for (int block_num = 0; block_num < blocks_req; block_num++) {
		//Allocate a vector to hold a block of shot_data
		std::vector<shotData> shot_block(file_block_size * num_gpu);

		//Populate the shot_block with data from file
		populateBlock(&shot_block, &filelist, block_num, num_gpu, file_block_size);

		//Sort tags and convert them to bins
		sortAndBinBlock(&shot_block, bin_width, num_gpu, file_block_size);
		//printShotChannelBins(&(shot_block[0]), 1);

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		/*cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
		}*/
		for (int gpu = 0; gpu < num_gpu; gpu++) {
			cudaStatus = hipSetDevice(gpu);
			if (cudaStatus != hipSuccess) {
				printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
				goto Error;
			}

			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				goto Error;
			}

			//Asyncronously load data to GPU
			for (int shot_file_num = 0; shot_file_num < file_block_size; shot_file_num++) {
				int block_file_num = shot_file_num * num_gpu + gpu;
				if ((shot_block)[block_file_num].file_load_completed) {
					int num_channels = (shot_block)[block_file_num].channel_list.size();
					if (num_channels >= 2) {


						std::vector<int32*> photon_bins;
						int32 start_and_end_clocks[2];
						std::vector<int32> photon_bins_length;
						photon_bins.resize(max_channels);
						photon_bins_length.resize(max_channels);

						start_and_end_clocks[0] = (shot_block)[block_file_num].sorted_clock_bins[1][0];
						start_and_end_clocks[1] = (shot_block)[block_file_num].sorted_clock_bins[0][0];
						for (int i = 0; i < num_channels; i++) {
							photon_bins[i] = &((shot_block)[block_file_num].sorted_photon_bins[i][0]);
							photon_bins_length[i] = (shot_block)[block_file_num].sorted_photon_tag_pointers[i];
						}

						//Synch to ensure previous asnyc memcopy has finished otherwise we'll start overwriting writing to data that may be DMA'd
						cudaStatus = hipEventSynchronize(events[gpu][shot_file_num]);
						if (cudaStatus != hipSuccess) {
							printf("Event synch failed\n");
							goto Error;
						}

						//Write photon bins to memory
						int photon_offset = shot_file_num * max_channels * max_tags_length;
						for (int i = 0; i < photon_bins_length.size(); i++) {
							memcpy(pinned_photon_bins[gpu] + photon_offset, (photon_bins)[i], (photon_bins_length)[i] * sizeof(int32));
							cudaStatus = hipMemcpyAsync((gpu_data[gpu]).photon_bins_gpu + photon_offset, pinned_photon_bins[gpu] + photon_offset, (photon_bins_length)[i] * sizeof(int32), hipMemcpyHostToDevice, streams[gpu][shot_file_num]);
							if (cudaStatus != hipSuccess) {
								printf("hipMemcpy photon_bins failed!\n");
								goto Error;
							}
							photon_offset += max_tags_length;
						}

						int clock_offset = shot_file_num * 2;
						//And other parameters
						memcpy(pinned_start_and_end_clocks[gpu] + clock_offset, start_and_end_clocks, 2 * sizeof(int32));
						cudaStatus = hipMemcpyAsync((gpu_data[gpu]).start_and_end_clocks_gpu + clock_offset, pinned_start_and_end_clocks[gpu] + clock_offset, 2 * sizeof(int32), hipMemcpyHostToDevice, streams[gpu][shot_file_num]);
						if (cudaStatus != hipSuccess) {
							printf("hipMemcpy clock_offset failed!\n");
							goto Error;
						}

						int length_offset = shot_file_num * max_channels;
						//Can't copy vector to cuda easily
						for (int i = 0; i < photon_bins_length.size(); i++) {
							memcpy(pinned_photon_bins_length[gpu] + i + length_offset, &((photon_bins_length)[i]), sizeof(int32));
						}
						cudaStatus = hipMemcpyAsync((gpu_data[gpu]).photon_bins_length_gpu + length_offset, pinned_photon_bins_length[gpu] + length_offset, max_channels * sizeof(int32), hipMemcpyHostToDevice, streams[gpu][shot_file_num]);
						if (cudaStatus != hipSuccess) {
							printf("hipMemcpy length_offset failed!\n");
							goto Error;
						}

						//Create an event to let us know all the async copies have occured
						hipEventRecord(events[gpu][shot_file_num], streams[gpu][shot_file_num]);
						//Run kernels
						calculateCoincidencesGPU_g2 << <cuda_blocks_numer, threads_per_cuda_block_numer, 0, streams[gpu][shot_file_num] >> > ((gpu_data[gpu]).coinc_gpu, (gpu_data[gpu]).photon_bins_gpu, (gpu_data[gpu]).start_and_end_clocks_gpu, (gpu_data[gpu]).max_bin_gpu, (gpu_data[gpu]).pulse_spacing_gpu, (gpu_data[gpu]).max_pulse_distance_gpu, (gpu_data[gpu]).offset_gpu, (gpu_data[gpu]).photon_bins_length_gpu, num_channels, shot_file_num);
					}
				}
			}
		}
		printf("Sent block %i/%i\n", block_num + 1, blocks_req);
	}
	printf("Finished sending blocks\n");

	for (int gpu = 0; gpu < num_gpu; gpu++) {
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}
		for (int i = 0; i < file_block_size; i++) {
			cudaStatus = hipStreamSynchronize(streams[gpu][i]);
			if (cudaStatus != hipSuccess) {
				printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
				goto Error;
			}
		}
	}

	//Free pinned memory
	for (int gpu = 0; gpu < num_gpu; gpu++) {
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}
		hipHostFree(pinned_photon_bins[gpu]);
		hipHostFree(pinned_photon_bins_length[gpu]);
		hipHostFree(pinned_start_and_end_clocks[gpu]);
	}

	//This is to pull the streamed numerator off the GPU
	//Streamed numerator refers to the way the numerator is stored on the GPU where each GPU stream has a seperate numerator
	for (int gpu = 0; gpu < num_gpu; gpu++) {
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}
		int32 *streamed_coinc;
		streamed_coinc = (int32 *)malloc(((2 * (max_bin)+1) + (max_pulse_distance * 2)) * file_block_size * sizeof(int32));

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(streamed_coinc, (gpu_data[gpu]).coinc_gpu, ((2 * (max_bin)+1) + (max_pulse_distance * 2)) * file_block_size * sizeof(int32), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			printf("hipMemcpy numerator failed!\n");
			free(streamed_coinc);
			goto Error;
		}

		//Collapse streamed coincidence counts down to regular numerator and denominator
		for (int i = 0; i < file_block_size; i++) {
			for (int j = 0; j < ((2 * (max_bin)+1) + (max_pulse_distance * 2)); j++) {
				if (j < (2 * (max_bin)+1)) {
					PyList_SetItem(numer, j, PyLong_FromLong(PyLong_AsLong(PyList_GetItem(numer, j)) + streamed_coinc[j + i * ((2 * (max_bin)+1) + (max_pulse_distance * 2))]));
				}
				else {
					denom[0] += streamed_coinc[j + i * ((2 * (max_bin)+1) + (max_pulse_distance * 2))];
				}
			}
		}
		free(streamed_coinc);
	}

	//Release CUDA device
	for (int gpu = 0; gpu < num_gpu; gpu++) {
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			printf("hipDeviceReset failed!\n");
		}
	}

Error:
	for (int gpu = 0; gpu < num_gpu; gpu++) {
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}
		hipFree((gpu_data[gpu].coinc_gpu));
		hipFree((gpu_data[gpu].offset_gpu));
		hipFree((gpu_data[gpu].max_bin_gpu));
		hipFree((gpu_data[gpu].pulse_spacing_gpu));
		hipFree((gpu_data[gpu].max_pulse_distance_gpu));
		hipFree((gpu_data[gpu].photon_bins_length_gpu));
		hipFree(gpu_data[gpu].photon_bins_gpu);
		hipFree(gpu_data[gpu].start_and_end_clocks_gpu);
		hipHostFree(pinned_photon_bins[gpu]);
		hipHostFree(pinned_photon_bins_length[gpu]);
		hipHostFree(pinned_start_and_end_clocks[gpu]);
		hipDeviceReset();
	}
}

DLLEXPORT void getG2Correlations_nostreams(char **file_list, int file_list_length, double max_time, double bin_width, double pulse_spacing, int max_pulse_distance, PyObject *numer, int32 *denom) {
	
	std::vector<char *> filelist(file_list_length);
	//Grab filename and stick it into filelist vector
	for (int i = 0; i < file_list_length; i++) {
		filelist[i] = file_list[i];
	}

	int blocks_req;
	if (file_list_length< (1 * num_gpu)) {
		blocks_req = 1;
	}
	else if ((file_list_length % (1 * num_gpu)) == 0) {
		blocks_req = file_list_length / (1 * num_gpu);
	}
	else {
		blocks_req = file_list_length / (1 * num_gpu) + 1;
	}
	printf("Chunking %i files into %i blocks\n", file_list_length, blocks_req);
	printf("Max Time\tBin Width\tPulse Spacing\tMax Pulse Distance\n");
	printf("%fus\t%fns\t%fus\t%i\n", max_time * 1e6, bin_width * 1e9, pulse_spacing * 1e6, max_pulse_distance);

	int max_bin = (int)round(max_time / bin_width);
	int bin_pulse_spacing = (int)round(pulse_spacing / bin_width);

	//Figure out how many blocks to chunk the processing up into
	//For the numerator
	int cuda_blocks_numer = 0;
	if (threads_per_cuda_block_numer >= (max_bin * 2 + 1) + (max_pulse_distance * 2)) {
		cuda_blocks_numer = 1;
	}
	else if (((max_bin * 2 + 1) % threads_per_cuda_block_numer) == 0) {
		cuda_blocks_numer = ((max_bin * 2 + 1) + (max_pulse_distance * 2)) / threads_per_cuda_block_numer;
	}
	else {
		cuda_blocks_numer = ((max_bin * 2 + 1) + (max_pulse_distance * 2)) / threads_per_cuda_block_numer + 1;
	}

	hipError_t cudaStatus;

	//Load some stuff to the GPU we will use permenantly
	//Allocate memory on GPU for various things
	gpuData gpu_data[num_gpu];
	
	for (int gpu = 0; gpu < num_gpu; gpu++) {

		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).photon_bins_gpu), max_channels * max_tags_length * 1 * sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc photon_bins_gpu failed\n");
			printf("%s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).offset_gpu), max_channels * 1 * sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc offset_gpu failed!\n");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).photon_bins_length_gpu), max_channels * 1 * sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc photon_bins_length_gpu failed!\n");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).coinc_gpu), ((2 * (max_bin)+1) + (max_pulse_distance * 2)) * 1 * sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc numer_gpu failed!\n");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).start_and_end_clocks_gpu), 2 * 1 * sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc start_and_end_clocks_gpu failed!\n");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).max_bin_gpu), sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc max_bin_gpu failed!\n");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).pulse_spacing_gpu), sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc pulse_spacing_gpu failed!\n");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).max_pulse_distance_gpu), sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc max_pulse_distance_gpu failed!\n");
			goto Error;
		}

		//And set some values that are constant across all data
		cudaStatus = hipMemcpy(((gpu_data[gpu]).max_bin_gpu), &max_bin, sizeof(int32), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			printf("hipMemcpy failed!\n");
			goto Error;
		}
		cudaStatus = hipMemcpy(((gpu_data[gpu]).pulse_spacing_gpu), &bin_pulse_spacing, sizeof(int32), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			printf("hipMemcpy failed!\n");
			goto Error;
		}
		cudaStatus = hipMemcpy(((gpu_data[gpu]).max_pulse_distance_gpu), &max_pulse_distance, sizeof(int32), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			printf("hipMemcpy failed!\n");
			goto Error;
		}

		//Pointer to first photon bin for each channel
		int host_offest_array[max_channels * 1];
		for (int i = 0; i < max_channels * 1; i++) {
			host_offest_array[i] = i * max_tags_length;
		}
		cudaStatus = hipMemcpy(((gpu_data[gpu]).offset_gpu), host_offest_array, max_channels * 1 * sizeof(int32), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			printf("hipMemcpy failed!\n");
			goto Error;
		}

		//Set numerator and denominator to 0
		cudaStatus = hipMemset(((gpu_data[gpu])).coinc_gpu, 0, ((2 * (max_bin)+1) + (max_pulse_distance * 2)) * 1 * sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMemset failed!\n");
			goto Error;
		}
	}

	//Processes files in blocks
	for (int block_num = 0; block_num < blocks_req; block_num++) {
		//Allocate a vector to hold a block of shot_data
		std::vector<shotData> shot_block(1 * num_gpu);

		//Populate the shot_block with data from file
		populateBlock(&shot_block, &filelist, block_num, num_gpu, 1);

		//Sort tags and convert them to bins
		sortAndBinBlock(&shot_block, bin_width, num_gpu, 1);
		//printShotChannelBins(&(shot_block[0]), 1);

		for (int gpu = 0; gpu < num_gpu; gpu++) {
			cudaStatus = hipSetDevice(gpu);
			if (cudaStatus != hipSuccess) {
				printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
				goto Error;
			}

			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				goto Error;
			}

			//Asyncronously load data to GPU
			for (int shot_file_num = 0; shot_file_num < 1; shot_file_num++) {
				int block_file_num = shot_file_num * num_gpu + gpu;
				if ((shot_block)[block_file_num].file_load_completed) {
					int num_channels = (shot_block)[block_file_num].channel_list.size();
					if (num_channels >= 2) {


						std::vector<int32*> photon_bins;
						int32 start_and_end_clocks[2];
						std::vector<int32> photon_bins_length;
						photon_bins.resize(max_channels);
						photon_bins_length.resize(max_channels);

						start_and_end_clocks[0] = (shot_block)[block_file_num].sorted_clock_bins[1][0];
						start_and_end_clocks[1] = (shot_block)[block_file_num].sorted_clock_bins[0][0];
						for (int i = 0; i < num_channels; i++) {
							photon_bins[i] = &((shot_block)[block_file_num].sorted_photon_bins[i][0]);
							photon_bins_length[i] = (shot_block)[block_file_num].sorted_photon_tag_pointers[i];
						}

						//Write photon bins to memory
						int photon_offset = shot_file_num * max_channels * max_tags_length;
						for (int i = 0; i < photon_bins_length.size(); i++) {
							cudaStatus = hipMemcpy((gpu_data[gpu]).photon_bins_gpu + photon_offset, (photon_bins)[i], (photon_bins_length)[i] * sizeof(int32), hipMemcpyHostToDevice);
							if (cudaStatus != hipSuccess) {
								printf("hipMemcpy photon_bins failed!\n");
								goto Error;
							}
							photon_offset += max_tags_length;
						}

						int clock_offset = shot_file_num * 2;
						//And other parameters
						cudaStatus = hipMemcpy((gpu_data[gpu]).start_and_end_clocks_gpu + clock_offset, start_and_end_clocks, 2 * sizeof(int32), hipMemcpyHostToDevice);
						if (cudaStatus != hipSuccess) {
							printf("hipMemcpy clock_offset failed!\n");
							goto Error;
						}

						int length_offset = shot_file_num * max_channels;
						//Can't copy vector to cuda easily
						for (int i = 0; i < photon_bins_length.size(); i++) {
							cudaStatus = hipMemcpy((gpu_data[gpu]).photon_bins_length_gpu + length_offset + i,  &((photon_bins_length)[i]), sizeof(int32), hipMemcpyHostToDevice);
							if (cudaStatus != hipSuccess) {
								printf("hipMemcpy length_offset failed!\n");
								goto Error;
							}
						}

						//Run kernels
						calculateCoincidencesGPU_g2_nominibatching << <cuda_blocks_numer, threads_per_cuda_block_numer, 0 >> > ((gpu_data[gpu]).coinc_gpu, (gpu_data[gpu]).photon_bins_gpu, (gpu_data[gpu]).start_and_end_clocks_gpu, (gpu_data[gpu]).max_bin_gpu, (gpu_data[gpu]).pulse_spacing_gpu, (gpu_data[gpu]).max_pulse_distance_gpu, (gpu_data[gpu]).offset_gpu, (gpu_data[gpu]).photon_bins_length_gpu, num_channels, shot_file_num);
					}
				}
			}
		}
		printf("Sent block %i/%i\n", block_num + 1, blocks_req);
	}
	printf("Finished sending blocks\n");

	for (int gpu = 0; gpu < num_gpu; gpu++) {
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}
	}

	//This is to pull the streamed numerator off the GPU
	//Streamed numerator refers to the way the numerator is stored on the GPU where each GPU stream has a seperate numerator
	for (int gpu = 0; gpu < num_gpu; gpu++) {
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}
		int32 *streamed_coinc;
		streamed_coinc = (int32 *)malloc(((2 * (max_bin)+1) + (max_pulse_distance * 2)) * 1 * sizeof(int32));

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(streamed_coinc, (gpu_data[gpu]).coinc_gpu, ((2 * (max_bin)+1) + (max_pulse_distance * 2)) * 1 * sizeof(int32), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			printf("hipMemcpy numerator failed!\n");
			free(streamed_coinc);
			goto Error;
		}

		//Collapse streamed coincidence counts down to regular numerator and denominator
		for (int i = 0; i < 1; i++) {
			for (int j = 0; j < ((2 * (max_bin)+1) + (max_pulse_distance * 2)); j++) {
				if (j < (2 * (max_bin)+1)) {
					PyList_SetItem(numer, j, PyLong_FromLong(PyLong_AsLong(PyList_GetItem(numer, j)) + streamed_coinc[j + i * ((2 * (max_bin)+1) + (max_pulse_distance * 2))]));
				}
				else {
					denom[0] += streamed_coinc[j + i * ((2 * (max_bin)+1) + (max_pulse_distance * 2))];
				}
			}
		}
		free(streamed_coinc);
	}

	//Release CUDA device
	for (int gpu = 0; gpu < num_gpu; gpu++) {
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			printf("hipDeviceReset failed!\n");
		}
	}

Error:
	for (int gpu = 0; gpu < num_gpu; gpu++) {
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}
		hipFree((gpu_data[gpu].coinc_gpu));
		hipFree((gpu_data[gpu].offset_gpu));
		hipFree((gpu_data[gpu].max_bin_gpu));
		hipFree((gpu_data[gpu].pulse_spacing_gpu));
		hipFree((gpu_data[gpu].max_pulse_distance_gpu));
		hipFree((gpu_data[gpu].photon_bins_length_gpu));
		hipFree(gpu_data[gpu].photon_bins_gpu);
		hipFree(gpu_data[gpu].start_and_end_clocks_gpu);
		hipDeviceReset();
	}
}

DLLEXPORT void getG2Correlations_nominibatching(char **file_list, int file_list_length, double max_time, double bin_width, double pulse_spacing, int max_pulse_distance, PyObject *numer, int32 *denom) {
	
	std::vector<char *> filelist(file_list_length);
	//Grab filename and stick it into filelist vector
	for (int i = 0; i < file_list_length; i++) {
		filelist[i] = file_list[i];
	}

	int blocks_req;
	if (file_list_length< (file_block_size * num_gpu)) {
		blocks_req = 1;
	}
	else if ((file_list_length % (file_block_size * num_gpu)) == 0) {
		blocks_req = file_list_length / (file_block_size * num_gpu);
	}
	else {
		blocks_req = file_list_length / (file_block_size * num_gpu) + 1;
	}
	printf("Chunking %i files into %i blocks\n", file_list_length, blocks_req);
	printf("Max Time\tBin Width\tPulse Spacing\tMax Pulse Distance\n");
	printf("%fus\t%fns\t%fus\t%i\n", max_time * 1e6, bin_width * 1e9, pulse_spacing * 1e6, max_pulse_distance);

	int max_bin = (int)round(max_time / bin_width);
	int bin_pulse_spacing = (int)round(pulse_spacing / bin_width);

	//Figure out how many blocks to chunk the processing up into
	//For the numerator
	int cuda_blocks_numer = 0;
	if (threads_per_cuda_block_numer >= (max_bin * 2 + 1) + (max_pulse_distance * 2)) {
		cuda_blocks_numer = 1;
	}
	else if (((max_bin * 2 + 1) % threads_per_cuda_block_numer) == 0) {
		cuda_blocks_numer = ((max_bin * 2 + 1) + (max_pulse_distance * 2)) / threads_per_cuda_block_numer;
	}
	else {
		cuda_blocks_numer = ((max_bin * 2 + 1) + (max_pulse_distance * 2)) / threads_per_cuda_block_numer + 1;
	}

	hipError_t cudaStatus;

	//Pointers for our various pinned memory for host-GPU DMA
	int32* pinned_photon_bins[num_gpu];
	int32* pinned_start_and_end_clocks[num_gpu];
	int32* pinned_photon_bins_length[num_gpu];

	//Load some stuff to the GPU we will use permenantly
	//Allocate memory on GPU for various things
	gpuData gpu_data[num_gpu];
	
	for (int gpu = 0; gpu < num_gpu; gpu++) {

		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		hipHostMalloc((int32**)&pinned_photon_bins[gpu], max_tags_length * max_channels * file_block_size * sizeof(int32));
		hipHostMalloc((int32**)&pinned_start_and_end_clocks[gpu], 2 * file_block_size * sizeof(int32));
		hipHostMalloc((int32**)&pinned_photon_bins_length[gpu], max_channels * file_block_size * sizeof(int32));

		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).photon_bins_gpu), max_channels * max_tags_length * file_block_size * sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc photon_bins_gpu failed\n");
			printf("%s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).offset_gpu), max_channels * file_block_size * sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc offset_gpu failed!\n");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).photon_bins_length_gpu), max_channels * file_block_size * sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc photon_bins_length_gpu failed!\n");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).coinc_gpu), ((2 * (max_bin)+1) + (max_pulse_distance * 2)) * file_block_size * sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc numer_gpu failed!\n");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).start_and_end_clocks_gpu), 2 * file_block_size * sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc start_and_end_clocks_gpu failed!\n");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).max_bin_gpu), sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc max_bin_gpu failed!\n");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).pulse_spacing_gpu), sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc pulse_spacing_gpu failed!\n");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&((gpu_data[gpu]).max_pulse_distance_gpu), sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMalloc max_pulse_distance_gpu failed!\n");
			goto Error;
		}

		//And set some values that are constant across all data
		cudaStatus = hipMemcpy(((gpu_data[gpu]).max_bin_gpu), &max_bin, sizeof(int32), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			printf("hipMemcpy failed!\n");
			goto Error;
		}
		cudaStatus = hipMemcpy(((gpu_data[gpu]).pulse_spacing_gpu), &bin_pulse_spacing, sizeof(int32), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			printf("hipMemcpy failed!\n");
			goto Error;
		}
		cudaStatus = hipMemcpy(((gpu_data[gpu]).max_pulse_distance_gpu), &max_pulse_distance, sizeof(int32), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			printf("hipMemcpy failed!\n");
			goto Error;
		}

		//Pointer to first photon bin for each channel
		int host_offest_array[max_channels * file_block_size];
		for (int i = 0; i < max_channels * file_block_size; i++) {
			host_offest_array[i] = i * max_tags_length;
		}
		cudaStatus = hipMemcpy(((gpu_data[gpu]).offset_gpu), host_offest_array, max_channels * file_block_size * sizeof(int32), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			printf("hipMemcpy failed!\n");
			goto Error;
		}

		//Set numerator and denominator to 0
		cudaStatus = hipMemset(((gpu_data[gpu])).coinc_gpu, 0, ((2 * (max_bin)+1) + (max_pulse_distance * 2)) * file_block_size * sizeof(int32));
		if (cudaStatus != hipSuccess) {
			printf("hipMemset failed!\n");
			goto Error;
		}
	}

	//Create some streams for us to use for GPU parallelism
	hipStream_t streams[num_gpu][file_block_size];
	for (int gpu = 0; gpu < num_gpu; gpu++) {
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}
		for (int i = 0; i < file_block_size; i++) {
			cudaStatus = hipStreamCreate(&streams[gpu][i]);
			if (cudaStatus != hipSuccess) {
				printf("Failed to create streams %s\n", hipGetErrorString(cudaStatus));
				goto Error;
			}
		}
	}

	//Create some events to allow us to know if previous transfer has completed
	hipEvent_t events[num_gpu][file_block_size];
	for (int gpu = 0; gpu < num_gpu; gpu++) {
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}
		for (int i = 0; i < file_block_size; i++) {
			hipEventCreate(&events[gpu][i]);
			if (cudaStatus != hipSuccess) {
				printf("Failed to create events %s\n", hipGetErrorString(cudaStatus));
				goto Error;
			}
		}
	}

	//Processes files in blocks
	for (int block_num = 0; block_num < blocks_req; block_num++) {
		//Allocate a vector to hold a block of shot_data
		std::vector<shotData> shot_block(file_block_size * num_gpu);

		//Populate the shot_block with data from file
		populateBlock(&shot_block, &filelist, block_num, num_gpu, file_block_size);

		//Sort tags and convert them to bins
		sortAndBinBlock(&shot_block, bin_width, num_gpu, file_block_size);
		//printShotChannelBins(&(shot_block[0]), 1);

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		/*cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
		}*/
		for (int gpu = 0; gpu < num_gpu; gpu++) {
			cudaStatus = hipSetDevice(gpu);
			if (cudaStatus != hipSuccess) {
				printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
				goto Error;
			}

			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				goto Error;
			}

			//Asyncronously load data to GPU
			for (int shot_file_num = 0; shot_file_num < file_block_size; shot_file_num++) {
				int block_file_num = shot_file_num * num_gpu + gpu;
				if ((shot_block)[block_file_num].file_load_completed) {
					int num_channels = (shot_block)[block_file_num].channel_list.size();
					if (num_channels >= 2) {


						std::vector<int32*> photon_bins;
						int32 start_and_end_clocks[2];
						std::vector<int32> photon_bins_length;
						photon_bins.resize(max_channels);
						photon_bins_length.resize(max_channels);

						start_and_end_clocks[0] = (shot_block)[block_file_num].sorted_clock_bins[1][0];
						start_and_end_clocks[1] = (shot_block)[block_file_num].sorted_clock_bins[0][0];
						for (int i = 0; i < num_channels; i++) {
							photon_bins[i] = &((shot_block)[block_file_num].sorted_photon_bins[i][0]);
							photon_bins_length[i] = (shot_block)[block_file_num].sorted_photon_tag_pointers[i];
						}

						//Synch to ensure previous asnyc memcopy has finished otherwise we'll start overwriting writing to data that may be DMA'd
						cudaStatus = hipEventSynchronize(events[gpu][shot_file_num]);
						if (cudaStatus != hipSuccess) {
							printf("Event synch failed\n");
							goto Error;
						}

						//Write photon bins to memory
						int photon_offset = shot_file_num * max_channels * max_tags_length;
						for (int i = 0; i < photon_bins_length.size(); i++) {
							memcpy(pinned_photon_bins[gpu] + photon_offset, (photon_bins)[i], (photon_bins_length)[i] * sizeof(int32));
							cudaStatus = hipMemcpyAsync((gpu_data[gpu]).photon_bins_gpu + photon_offset, pinned_photon_bins[gpu] + photon_offset, (photon_bins_length)[i] * sizeof(int32), hipMemcpyHostToDevice, streams[gpu][shot_file_num]);
							if (cudaStatus != hipSuccess) {
								printf("hipMemcpy photon_bins failed!\n");
								goto Error;
							}
							photon_offset += max_tags_length;
						}

						int clock_offset = shot_file_num * 2;
						//And other parameters
						memcpy(pinned_start_and_end_clocks[gpu] + clock_offset, start_and_end_clocks, 2 * sizeof(int32));
						cudaStatus = hipMemcpyAsync((gpu_data[gpu]).start_and_end_clocks_gpu + clock_offset, pinned_start_and_end_clocks[gpu] + clock_offset, 2 * sizeof(int32), hipMemcpyHostToDevice, streams[gpu][shot_file_num]);
						if (cudaStatus != hipSuccess) {
							printf("hipMemcpy clock_offset failed!\n");
							goto Error;
						}

						int length_offset = shot_file_num * max_channels;
						//Can't copy vector to cuda easily
						for (int i = 0; i < photon_bins_length.size(); i++) {
							memcpy(pinned_photon_bins_length[gpu] + i + length_offset, &((photon_bins_length)[i]), sizeof(int32));
						}
						cudaStatus = hipMemcpyAsync((gpu_data[gpu]).photon_bins_length_gpu + length_offset, pinned_photon_bins_length[gpu] + length_offset, max_channels * sizeof(int32), hipMemcpyHostToDevice, streams[gpu][shot_file_num]);
						if (cudaStatus != hipSuccess) {
							printf("hipMemcpy length_offset failed!\n");
							goto Error;
						}

						//Create an event to let us know all the async copies have occured
						hipEventRecord(events[gpu][shot_file_num], streams[gpu][shot_file_num]);
						//Run kernels
						calculateCoincidencesGPU_g2_nominibatching << <cuda_blocks_numer, threads_per_cuda_block_numer, 0, streams[gpu][shot_file_num] >> > ((gpu_data[gpu]).coinc_gpu, (gpu_data[gpu]).photon_bins_gpu, (gpu_data[gpu]).start_and_end_clocks_gpu, (gpu_data[gpu]).max_bin_gpu, (gpu_data[gpu]).pulse_spacing_gpu, (gpu_data[gpu]).max_pulse_distance_gpu, (gpu_data[gpu]).offset_gpu, (gpu_data[gpu]).photon_bins_length_gpu, num_channels, shot_file_num);
					}
				}
			}
		}
		printf("Sent block %i/%i\n", block_num + 1, blocks_req);
	}
	printf("Finished sending blocks\n");

	for (int gpu = 0; gpu < num_gpu; gpu++) {
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}
		for (int i = 0; i < file_block_size; i++) {
			cudaStatus = hipStreamSynchronize(streams[gpu][i]);
			if (cudaStatus != hipSuccess) {
				printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
				goto Error;
			}
		}
	}

	//Free pinned memory
	for (int gpu = 0; gpu < num_gpu; gpu++) {
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}
		hipHostFree(pinned_photon_bins[gpu]);
		hipHostFree(pinned_photon_bins_length[gpu]);
		hipHostFree(pinned_start_and_end_clocks[gpu]);
	}

	//This is to pull the streamed numerator off the GPU
	//Streamed numerator refers to the way the numerator is stored on the GPU where each GPU stream has a seperate numerator
	for (int gpu = 0; gpu < num_gpu; gpu++) {
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}
		int32 *streamed_coinc;
		streamed_coinc = (int32 *)malloc(((2 * (max_bin)+1) + (max_pulse_distance * 2)) * file_block_size * sizeof(int32));

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(streamed_coinc, (gpu_data[gpu]).coinc_gpu, ((2 * (max_bin)+1) + (max_pulse_distance * 2)) * file_block_size * sizeof(int32), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			printf("hipMemcpy numerator failed!\n");
			free(streamed_coinc);
			goto Error;
		}

		//Collapse streamed coincidence counts down to regular numerator and denominator
		for (int i = 0; i < file_block_size; i++) {
			for (int j = 0; j < ((2 * (max_bin)+1) + (max_pulse_distance * 2)); j++) {
				if (j < (2 * (max_bin)+1)) {
					PyList_SetItem(numer, j, PyLong_FromLong(PyLong_AsLong(PyList_GetItem(numer, j)) + streamed_coinc[j + i * ((2 * (max_bin)+1) + (max_pulse_distance * 2))]));
				}
				else {
					denom[0] += streamed_coinc[j + i * ((2 * (max_bin)+1) + (max_pulse_distance * 2))];
				}
			}
		}
		free(streamed_coinc);
	}

	//Release CUDA device
	for (int gpu = 0; gpu < num_gpu; gpu++) {
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			printf("hipDeviceReset failed!\n");
		}
	}

Error:
	for (int gpu = 0; gpu < num_gpu; gpu++) {
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}
		hipFree((gpu_data[gpu].coinc_gpu));
		hipFree((gpu_data[gpu].offset_gpu));
		hipFree((gpu_data[gpu].max_bin_gpu));
		hipFree((gpu_data[gpu].pulse_spacing_gpu));
		hipFree((gpu_data[gpu].max_pulse_distance_gpu));
		hipFree((gpu_data[gpu].photon_bins_length_gpu));
		hipFree(gpu_data[gpu].photon_bins_gpu);
		hipFree(gpu_data[gpu].start_and_end_clocks_gpu);
		hipHostFree(pinned_photon_bins[gpu]);
		hipHostFree(pinned_photon_bins_length[gpu]);
		hipHostFree(pinned_start_and_end_clocks[gpu]);
		hipDeviceReset();
	}
}

DLLEXPORT void getG2Correlations_cpu(char **file_list, int file_list_length, double max_time, double bin_width, double pulse_spacing, int max_pulse_distance, PyObject *numer, int32 *denom, int num_cpu_threads) {


	std::vector<char *> filelist(file_list_length);
	//Grab filename and stick it into filelist vector
	for (int i = 0; i < file_list_length; i++) {
		filelist[i] = file_list[i];
	}

	int max_bin = (int)round(max_time / bin_width);
	int bin_pulse_spacing = (int)round(pulse_spacing / bin_width);

	int32 *coinc;
	coinc = (int32*)malloc(((2 * (max_bin)+1) + (max_pulse_distance * 2)) * num_cpu_threads * sizeof(int32));

	for (int id = 0; id < ((2 * (max_bin)+1) + (max_pulse_distance * 2)) * num_cpu_threads; id++) {
		coinc[id] = 0;
	}

	int blocks_req;
	if (file_list_length < (num_cpu_threads)) {
		blocks_req = 1;
	}
	else if ((file_list_length % (num_cpu_threads)) == 0) {
		blocks_req = file_list_length / (num_cpu_threads);
	}
	else {
		blocks_req = file_list_length / (num_cpu_threads) + 1;
	}

	printf("Chunking %i files into %i blocks\n", file_list_length, blocks_req);
	printf("Max Time\tBin Width\tPulse Spacing\tMax Pulse Distance\n");
	printf("%fus\t%fns\t%fus\t%i\n", max_time * 1e6, bin_width * 1e9, pulse_spacing * 1e6, max_pulse_distance);

	//Processes files in blocks
	for (int block_num = 0; block_num < blocks_req; block_num++) {
		//Allocate a vector to hold a block of shot_data
		std::vector<shotData> shot_block(num_cpu_threads);

		//Populate the shot_block with data from file
		populateBlock(&shot_block, &filelist, block_num, 1, num_cpu_threads);

		//Sort tags and convert them to bins
		sortAndBinBlock(&shot_block, bin_width, 1, num_cpu_threads);

		//Processes files
		#pragma omp parallel for num_threads(num_cpu_threads)
		for (int shot_file_num = 0; shot_file_num < num_cpu_threads; shot_file_num++) {
			if ((shot_block)[shot_file_num].file_load_completed) {

				#pragma omp parallel for
				for (int32 id = 0; id < (2 * (max_bin)+1) + (max_pulse_distance * 2); id++) {
					calculateCoincidencesGPU_g2_cpu(&(shot_block[shot_file_num]), id, &max_bin, &bin_pulse_spacing, &max_pulse_distance, coinc, shot_file_num);
				}
			}
		}

	}

	//Collapse streamed coincidence counts down to regular numerator and denominator
	for (int i = 0; i < num_cpu_threads; i++) {
		for (int j = 0; j < ((2 * (max_bin)+1) + (max_pulse_distance * 2)); j++) {
			if (j < (2 * (max_bin)+1)) {
				PyList_SetItem(numer, j, PyLong_FromLong(PyLong_AsLong(PyList_GetItem(numer, j)) + coinc[j + i * ((2 * (max_bin)+1) + (max_pulse_distance * 2))]));
			}
			else {
				denom[0] += coinc[j + i * ((2 * (max_bin)+1) + (max_pulse_distance * 2))];
			}
		}
	}
	free(coinc);

}

DLLEXPORT void getG2Correlations_cpu_thread_benchmark(char **file_list, int file_list_length, double max_time, double bin_width, double pulse_spacing, int max_pulse_distance, PyObject *numer, int32 *denom, int num_cpu_threads) {


	std::vector<char *> filelist(file_list_length);
	//Grab filename and stick it into filelist vector
	for (int i = 0; i < file_list_length; i++) {
		filelist[i] = file_list[i];
	}

	int max_bin = (int)round(max_time / bin_width);
	int bin_pulse_spacing = (int)round(pulse_spacing / bin_width);

	int32 *coinc;
	coinc = (int32*)malloc(((2 * (max_bin)+1) + (max_pulse_distance * 2)) * 32 * sizeof(int32));

	for (int id = 0; id < ((2 * (max_bin)+1) + (max_pulse_distance * 2)) * 32; id++) {
		coinc[id] = 0;
	}

	int blocks_req = file_list_length;
	if (file_list_length < (32)) {
		blocks_req = 1;
	}
	else if ((file_list_length % (32)) == 0) {
		blocks_req = file_list_length / (32);
	}
	else {
		blocks_req = file_list_length / (32)+1;
	}

	printf("Chunking %i files into %i blocks\n", file_list_length, blocks_req);
	printf("Max Time\tBin Width\tPulse Spacing\tMax Pulse Distance\n");
	printf("%fus\t%fns\t%fus\t%i\n", max_time * 1e6, bin_width * 1e9, pulse_spacing * 1e6, max_pulse_distance);
	printf("Using %i CPU threads\n", num_cpu_threads);

	//Processes files in blocks
	for (int block_num = 0; block_num < blocks_req; block_num++) {
		//Allocate a vector to hold a block of shot_data
		std::vector<shotData> shot_block(32);

		//Populate the shot_block with data from file
		populateBlock(&shot_block, &filelist, block_num, 1, 32);

		//Sort tags and convert them to bins
		sortAndBinBlock(&shot_block, bin_width, 1, 32);

		//Processes files
		for (int shot_file_num = 0; shot_file_num < 32; shot_file_num++) {
			if ((shot_block)[shot_file_num].file_load_completed) {
				#pragma omp parallel for num_threads(num_cpu_threads)
				for (int32 id = 0; id < (2 * (max_bin)+1) + (max_pulse_distance * 2); id++) {
					calculateCoincidencesGPU_g2_cpu(&(shot_block[shot_file_num]), id, &max_bin, &bin_pulse_spacing, &max_pulse_distance, coinc, shot_file_num);
				}
			}
		}

	}

	//Collapse streamed coincidence counts down to regular numerator and denominator
	for (int i = 0; i < 32; i++) {
		for (int j = 0; j < ((2 * (max_bin)+1) + (max_pulse_distance * 2)); j++) {
			if (j < (2 * (max_bin)+1)) {
				PyList_SetItem(numer, j, PyLong_FromLong(PyLong_AsLong(PyList_GetItem(numer, j)) + coinc[j + i * ((2 * (max_bin)+1) + (max_pulse_distance * 2))]));
			}
			else {
				denom[0] += coinc[j + i * ((2 * (max_bin)+1) + (max_pulse_distance * 2))];
			}
		}
	}
	free(coinc);

}
